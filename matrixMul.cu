#include "hip/hip_runtime.h"
/*lab-3 of ECE 285 GPU Programming
 Student: Wenyu Zhang
 PID: A53238371
 Email: wez078@ucsd.edu*/

#include <cstring>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <string>
#include <iomanip>
#include <stdexcept>
#include <vector>
#include <numeric>
// CPU library for Half float
#include "./half-1.12.0/half.hpp"

#include <algorithm>
#include <cctype>
#include <cmath>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "tmm.h"
using namespace std;
using namespace tmm;
// k128-w32
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <tuple>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <sstream>
#include <math.h>
#if defined USEOMP
#include <omp.h>
#endif

//func declarations
__global__ void tmm_kernel(Parameter para, long long a_seg, long long b_seg, tmm_int k, half *gpuHalfp, half *gpuHalfq, half *gpuR);
__global__ void transform_half(half *gpu_half_feature, float *gpu_float_feature, long long vec_size);
tmm_double tmm_update_k128(Parameter para, tmm_model *model, tmm_problem *prob);
tmm_problem read_problem(string path);
void grid_problem(tmm_problem* prob, Parameter para);
tmm_model* tmm_load_model(char const *path);
void init_model(tmm_model*model, tmm_problem*prob, Parameter para);
void tmm_destroy_model(tmm_model **model);
void transform_R(short *halfR, float *floatR, tmm_problem *prob, Parameter para);
tmm_float look_up_floatR(tmm_int u, tmm_int v, tmm_model *model, tmm_problem *prob);
tmm_double calc_rmse(tmm_problem *prob, tmm_model *model, int rn, int cn);

//------------------------------------------------------------Kernel Functions-----------------------------------------------

//-------------------------kernel----------------------------
__global__ void tmm_kernel(Parameter para, long long a_seg, long long b_seg, tmm_int k, half *gpuHalfp, half *gpuHalfq, half *gpuR)
{
    //from GMEM to SMEM
    int bx = blockDim.x, by = blockDim.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int I = blockIdx.x*bx + tx, J = blockIdx.y*by + ty; // row and col
    int t = para.tile;
	__shared__ half aTile[32][32];
	__shared__ half bTile[32][32];
    float c = 0.0f;     
	for (unsigned int k1=0; k1 < k/t+1; k1++)
	{
		if (I < a_seg && k1*t+ty < k){
			aTile[tx][ty] = __ldg(&gpuHalfp[I*k + k1*t + ty]);
		}
		else aTile[tx][ty] = __float2half(0.0);
		if (J < b_seg && k1*t+tx < k){
			bTile[tx][ty] = __ldg(&gpuHalfq[J*k + k1*t + tx]);
		}
		else bTile[tx][ty] = __float2half(0.0);
		__syncthreads(); // Synchronizes all threads in a block	
		for (unsigned int k2=0; k2< bx; k2++)
            c += __half2float(aTile[tx][k2])*__half2float(bTile[k2][ty]);
        __syncthreads(); // Avoids memory hazards
	}
	if (I < a_seg && J < b_seg)
		gpuR[I*b_seg + J] = __float2half(c);// __float2half(c);
}
//--------------------------------------------------------------------
__global__ void transform_half(half *gpu_half_feature, float *gpu_float_feature, long long vec_size)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int number_threads = gridDim.x*blockDim.x;

	for (long long i = tid; i < vec_size; i += number_threads)
	{
		gpu_float_feature[i] = __half2float(gpu_half_feature[i]);
	}
}
//---------------------------------------------------------------------
void transform_R(short *halfR, float *floatR, tmm_problem *prob, Parameter para)
{

	half *gpu_half_R;
	float *gpu_float_R;

	hipMalloc(&gpu_half_R, sizeof(half)*prob->gridSizeM*prob->gridSizeN);
	hipMalloc(&gpu_float_R, sizeof(float)*prob->gridSizeM*prob->gridSizeN);
	//gpuErr(hipPeekAtLastError());

	hipMemcpy(gpu_half_R, halfR, sizeof(half)*prob->gridSizeM*prob->gridSizeN, hipMemcpyHostToDevice);
	//gpuErr(hipPeekAtLastError());

	int num_blocks = (prob->gridSizeM*prob->gridSizeN + 255) / 256;
	if (num_blocks > 8 * 24)num_blocks = 8 * 24;

	transform_half <<<num_blocks, 256 >>>(gpu_half_R, gpu_float_R, prob->gridSizeM*prob->gridSizeN);

	//gpuErr(hipPeekAtLastError());
	hipMemcpy(floatR, gpu_float_R, sizeof(float)*prob->gridSizeM*prob->gridSizeN, hipMemcpyDeviceToHost);
	//gpuErr(hipPeekAtLastError());

	hipFree(gpu_half_R);
	hipFree(gpu_float_R);
	//gpuErr(hipPeekAtLastError());
}

//--------------------------core---------------------------------------

tmm_double tmm_update_k128(Parameter para, tmm_model *model, tmm_problem *prob)
{   using half_float::half;
    printf("calling tmm_update_k128()...\n");
	printf("gridSizeM: %d\n", model->gridSizeM);
	printf("gridSizeN: %d\n", model->gridSizeN);

	printf("k: %d\n", model->k);
    //malloc
    hipMalloc(reinterpret_cast<void**>(&model->gpuHalfp), sizeof(half)*model->gridSizeM*model->k);
    hipMalloc(reinterpret_cast<void**>(&model->gpuHalfq), sizeof(half)*model->gridSizeN*model->k);
	hipMalloc(reinterpret_cast<void**>(&prob->gpuR), sizeof(half)*model->gridSizeM*model->gridSizeN);
	
	//gpuErr(hipPeekAtLastError());
	tmm_double loss = 0;
	for (int rowTile = 0; rowTile < para.rowScale; rowTile++)
	{
		for (int colTile = 0; colTile < para.colScale; colTile++)
		{
			short *p_tmp = model->halfp + model->gridSizeM*model->k*rowTile;
            short *q_tmp = model->halfq + model->gridSizeN*model->k*colTile;
			assert(p_tmp);
			assert(q_tmp); 
			
			// Copy from CPU to GMEM
			hipMemcpy(model->gpuHalfp, p_tmp, sizeof(half)*model->gridSizeM*model->k, hipMemcpyHostToDevice);
            hipMemcpy(model->gpuHalfq, q_tmp, sizeof(half)*model->gridSizeN*model->k, hipMemcpyHostToDevice);
			
			// Dim Configuration
            dim3 block(32, 32);
			dim3 grid((model->gridSizeM+block.x-1)/block.x, (model->gridSizeN+block.y-1)/block.y);
			// Kernel
			tmm_kernel<<<grid, block>>>(para, model->gridSizeM, model->gridSizeN, model->k, model->gpuHalfp, model->gpuHalfq, prob->gpuR);
			// Copy from GMEM to CPU
			//short *R_tmp = prob->halfR + model->gridSizeM*model->gridSizeN*(para.colScale*rowTile+colTile);
			hipMemcpy(prob->halfR, prob->gpuR, (long long)(sizeof(half))*model->gridSizeM*model->gridSizeN , hipMemcpyDeviceToHost);
            transform_R(prob->halfR, prob->floatR, prob, para);
			printf("calculating rmse ... gridx: %d, gridy: %d\n", rowTile, colTile);
			loss += calc_rmse(prob, model, rowTile, colTile);
		}
	}
    hipDeviceSynchronize();

    hipFree(model->gpuHalfp);
    hipFree(model->gpuHalfq);
	hipFree(prob->gpuR);
	return sqrt(loss / prob->nnz);
}
//------------------------------------------------------------Host Functions-------------------------------------------------
//----------------------------------------------------------
tmm_problem read_problem(string path)  //load matrix R(m n nnz R)
{
	//A simple function that reads the sparse matrix in COO manner.
	printf("read_problem:%s\n", path.c_str());
	tmm_problem prob;
	prob.m = 1;
	prob.n = 1;
	prob.nnz = 0;
	prob.R = nullptr;


	if (path.empty())
		return prob;

	FILE*fptr = fopen(path.c_str(), "rb");
	if (fptr == NULL) {
		printf("error file open %s\n", path.c_str());
		return prob;
	}


	unsigned int tmp;
	fread(&prob.m, sizeof(unsigned int), 1, fptr);
	fread(&prob.n, sizeof(unsigned int), 1, fptr);
	fread(&tmp, sizeof(unsigned int), 1, fptr);
	prob.nnz = tmp;

	tmm_node *R = new tmm_node[prob.nnz];


	long long idx = 0;
	while (true)
	{
		int flag = 0;
		int u, v;
		float r;

		flag += fread(&u, sizeof(int), 1, fptr);
		flag += fread(&v, sizeof(int), 1, fptr);
		flag += fread(&r, sizeof(float), 1, fptr);

		if (flag != 3)break;

		R[idx].u = u;
		R[idx].v = v;
		R[idx].r = r;
		idx++;
	}
	prob.R = R;

	fclose(fptr);
	printf("m:%lld, n:%lld, nnz:%lld\n\n", prob.m, prob.n, prob.nnz);
	return prob;
}
//-----------------------------------------------------------------
void grid_problem(tmm_problem* prob, Parameter para) //grid matrix R(gridSizeM, gridSizeN, halfR, floatR)
{
	clock_t start;

	printf("grid problem ...\n\n\n");
	fflush(stdout);

	//grid the problem into several grids
	long long u_seg, v_seg;
	u_seg = (long long)ceil((double)prob->m / para.rowScale);
	v_seg = (long long)ceil((double)prob->n / para.colScale);

	prob->gridSizeM = u_seg;
	prob->gridSizeN = v_seg;

	prob->halfR = (short*)malloc((long long)(sizeof(short)) * prob->gridSizeM * prob->gridSizeN);
    prob->floatR = (float*)malloc((long long)(sizeof(float)) * prob->gridSizeM * prob->gridSizeN);
}

//-------------------------------------------
tmm_model* tmm_load_model(char const *path)  // load feature matrix P, Q(m, n, k, P, Q)
{
	printf("tmm_load_model called\n");

	FILE* fptr = fopen(path, "rb");
	if (fptr == NULL)
	{
		printf("%s open failed\n", path);
		exit(0);
	}
	clock_t start = clock();

	tmm_model *model = new tmm_model;
	model->P = nullptr;
	model->Q = nullptr;

	int count;

	int tmp_f, tmp_m, tmp_n, tmp_k;

	count = fread(&tmp_m, sizeof(int), 1, fptr);
	count = fread(&tmp_n, sizeof(int), 1, fptr);
	count = fread(&tmp_k, sizeof(int), 1, fptr);

	model->m = tmp_m;
	model->n = tmp_n;
	model->k = tmp_k;

	printf("m:   %lld\n", model->m);
	printf("n:   %lld\n", model->n);
	printf("k:   %lld\n", model->k);

	printf("p_size:%lld\n", ((long long)model->m)*model->k);

	try
	{
		model->P = malloc_aligned_float<short>((tmm_long)model->m*model->k);
		model->Q = malloc_aligned_float<short>((tmm_long)model->n*model->k);
	}
	catch (bad_alloc const &e)
	{
		cerr << e.what() << endl;
		tmm_destroy_model(&model);
		return nullptr;
	}

	auto read = [&](short *ptr, tmm_int size)
	{
		for (tmm_int i = 0; i < size; i++)
		{
			short *ptr1 = ptr + (tmm_long)i*model->k;
			count = fread(ptr1, sizeof(short), model->k, fptr);
			if (i % 100000000 == 0)printf("progress:%%%.3f\n", ((double)100.0)*i / size);
		}
	};


	printf("loading feature p m:%lld ...\n", model->m);
	read(model->P, model->m);
	printf("loading feature q n:%lld ...\n", model->n);
	read(model->Q, model->n);

	printf("time elapsed:%.8lfs\n\n", (clock() - start) / (double)CLOCKS_PER_SEC);

	return model;
}
//-----------------------------------------------------------------
void init_model(tmm_model*model, tmm_problem*prob, Parameter para)//malloc(gridSizeM, gridSizeN, halfpq)
{

    printf("init model ...\n");
    clock_t start = clock();

    //tmm_model *model = new tmm_model;
    //model->fun = 0;
    model->gridSizeM = prob->gridSizeM;
    model->gridSizeN = prob->gridSizeN;

    //allocate memory
    hipHostMalloc(&model->halfp, sizeof(short)*model->gridSizeM*para.rowScale*model->k);
    hipHostMalloc(&model->halfq, sizeof(short)*model->gridSizeN*para.colScale*model->k);
	hipMemcpy(model->halfp, model->P, sizeof(short)*model->m*model->k, hipMemcpyHostToHost);
	hipMemcpy(model->halfq, model->Q, sizeof(short)*model->n*model->k, hipMemcpyHostToHost);
	for (long long i = model->m; i < model->gridSizeM*para.rowScale; i++) {
		model->halfp[i] = 0;
	}
	for (long long i = model->n; i < model->gridSizeN*para.colScale; i++) {
		model->halfq[i] = 0;
	}
    printf("time elapsed:%.8lfs\n",(clock() - start)/(double)CLOCKS_PER_SEC);
    printf("\n\n\n");
}

//-----------------------------------------
void tmm_destroy_model(tmm_model **model)
{
	if (model == nullptr || *model == nullptr)
		return;
#ifdef _WIN32
	_aligned_free((*model)->P);
	_aligned_free((*model)->Q);
#else
	free((*model)->P);
	free((*model)->Q);
#endif
	delete *model;
	*model = nullptr;
}

//-----------------------------------------------------------------
tmm_float look_up_floatR(tmm_int u, tmm_int v, tmm_model *model, tmm_problem *prob)
{
	int gridm = u%model->gridSizeM;
	int gridn = v%model->gridSizeN;
	int id = gridm * model->gridSizeN + gridn;
	return prob->floatR[id];

}
//-------------------------------------------------------
tmm_double calc_rmse(tmm_problem *prob, tmm_model *model, int rn, int cn)
{

	if (prob->nnz == 0)
		return 0;
	tmm_double loss = 0;

	for (tmm_long i = 0; i < prob->nnz; i++)
	{   
		tmm_node &N = prob->R[i];

		if ( (model->gridSizeM*rn <= N.u) && (N.u < model->gridSizeM*(rn+1)) && (model->gridSizeN*cn <= N.v) && (N.v < model->gridSizeN*(cn + 1)) ){			
		    N.rp = look_up_floatR(N.u, N.v, model, prob);
			//printf("rp=%f\n", N.rp);
		    tmm_float e = N.r - look_up_floatR(N.u, N.v, model, prob);
		    loss += e*e;
		}

		if (i % 100000000 == 0 && i > 0)printf("progress: %%%.3lf, est_RMSE: %.4lf\n", ((double)100.0)*i / prob->nnz, sqrt(loss / (i + 1))); 
	}
	return loss;
}
//----------------multiplication-----------------------------
void multiplication(string test_path, const char* model_path)
{
    Parameter para;
    tmm_problem prob = read_problem(test_path);  //"netflix_mme.bin" "netflix_mm.bin" 
    grid_problem(&prob, para);
    tmm_model *model = tmm_load_model(model_path); //"pqmodel_hf.bin"
    if(model == nullptr)
        throw runtime_error("cannot load model from " + string(model_path));
    init_model(model, &prob, para);

    //core
	auto rmse = tmm_update_k128(para, model, &prob);

	cout << fixed << setprecision(4) << "RMSE = " << rmse << endl;

    tmm_destroy_model(&model);
}
//----------------------------------------------------------
#define MByte (1024∗1024) 

int main()
{
	string test_path = "C:/Users/wez078/lab3/Src/lab3/netflix_mme.bin";
	const char* model_path = "C:/Users/wez078/lab3/Src/lab3/pqmodel_hf.bin";

    multiplication(test_path, model_path);

    return 0;
}
